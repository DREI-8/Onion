#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdexcept>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>

bool is_cuda_available() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return (error == hipSuccess && deviceCount > 0);
}

__host__ void cpu_to_cuda(Tensor* tensor) {
    if (tensor->device && strcmp(tensor->device.get(), "cuda") == 0) {
        return;
    }

    float* device_data;
    hipError_t error = hipMalloc((void**)&device_data, tensor->size * sizeof(float));
    
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::runtime_error("Failed to allocate GPU memory");
    }

    error = hipMemcpy(device_data, tensor->data.get(), tensor->size * sizeof(float), hipMemcpyHostToDevice);
    
    if (error != hipSuccess) {
        hipFree(device_data);
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::runtime_error("Failed to copy data to GPU");
    }

    auto cuda_deleter = [](float* ptr) { hipFree(ptr); };

    tensor->data = std::shared_ptr<float[]>(device_data, cuda_deleter);

    const char* device_str = "cuda";
    size_t str_len = strlen(device_str) + 1;
    tensor->device = std::shared_ptr<char[]>(strdup(device_str), [](char* p) { free(p); });
}

__host__ void cuda_to_cpu(Tensor* tensor) {
    if (!tensor->device || strcmp(tensor->device.get(), "cpu") == 0) {
        return;
    }

    float* host_data = new float[tensor->size];

    hipError_t error = hipMemcpy(host_data, tensor->data.get(), tensor->size * sizeof(float), hipMemcpyDeviceToHost);
    
    if (error != hipSuccess) {
        delete[] host_data;
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::runtime_error("Failed to copy data from GPU");
    }

    tensor->data = std::shared_ptr<float[]>(host_data);

    const char* device_str = "cpu";
    size_t str_len = strlen(device_str) + 1;
    tensor->device = std::shared_ptr<char[]>(strdup(device_str), [](char* p) { free(p); });
}

void to_device(Tensor* tensor, const char* target_device) {
    const char* current_device = tensor->device ? tensor->device.get() : "cpu";

    if (strcmp(current_device, target_device) == 0) {
        return;
    }
    
    if (strcmp(target_device, "cuda") == 0) {
        cpu_to_cuda(tensor);
    } 
    else if (strcmp(target_device, "cpu") == 0) {
        cuda_to_cpu(tensor);
    } 
    else {
        fprintf(stderr, "Unsupported device: %s\n", target_device);
        throw std::runtime_error("Unsupported device");
    }
}

__global__ void add_kernel(const float* a, const float* b, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] + b[idx];
    }
}

Tensor add_tensor_cuda(const Tensor& a, const Tensor& b) {
    if (a.size != b.size) {
        throw std::runtime_error("Tensors must have same size for CUDA addition");
    }

    float* result_data;
    hipMalloc(&result_data, a.size * sizeof(float));

    int block_size = 256;
    int num_blocks = (a.size + block_size - 1) / block_size;
    add_kernel<<<num_blocks, block_size>>>(a.data.get(), b.data.get(), result_data, a.size);
    hipDeviceSynchronize();

    int* shape_copy = new int[a.ndim];
    memcpy(shape_copy, a.shape.get(), a.ndim * sizeof(int));

    auto cuda_deleter = [](float* ptr) { hipFree(ptr); };
    std::shared_ptr<float[]> shared_result(result_data, cuda_deleter);
    
    Tensor result(shared_result, shape_copy, a.ndim);
    
    const char* device_str = "cuda";
    size_t str_len = strlen(device_str) + 1;
    result.device = std::shared_ptr<char[]>(strdup("cuda"), [](char* p) { delete[] p; });
    
    return result;
}

__global__ void subtract_kernel(const float* a, const float* b, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] - b[idx];
    }
}

Tensor sub_tensor_cuda(const Tensor& a, const Tensor& b) {
    if (a.size != b.size) {
        throw std::runtime_error("Tensors must have same size for CUDA subtraction");
    }

    float* result_data;
    hipMalloc(&result_data, a.size * sizeof(float));

    int block_size = 256;
    int num_blocks = (a.size + block_size - 1) / block_size;
    subtract_kernel<<<num_blocks, block_size>>>(a.data.get(), b.data.get(), result_data, a.size);
    hipDeviceSynchronize();

    int* shape_copy = new int[a.ndim];
    memcpy(shape_copy, a.shape.get(), a.ndim * sizeof(int));

    auto cuda_deleter = [](float* ptr) { hipFree(ptr); };
    std::shared_ptr<float[]> shared_result(result_data, cuda_deleter);
    
    Tensor result(shared_result, shape_copy, a.ndim);
    result.device = std::shared_ptr<char[]>(strdup("cuda"), [](char* p) { delete[] p; });
    
    return result;
}

#else

bool is_cuda_available() {
    return false;
}

void cpu_to_cuda(Tensor* tensor) {
    fprintf(stderr, "CUDA not available in this build\n");
    throw std::runtime_error("CUDA not available");
}

void cuda_to_cpu(Tensor* tensor) {
    fprintf(stderr, "CUDA not available in this build\n");
    throw std::runtime_error("CUDA not available");
}

void to_device(Tensor* tensor, const char* target_device) {
    if (strcmp(target_device, "cuda") == 0) {
        fprintf(stderr, "CUDA not available in this build\n");
        throw std::runtime_error("CUDA not available");
    }
}

#endif // __HIPCC__