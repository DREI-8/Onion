#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdexcept>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>

bool is_cuda_available() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return (error == hipSuccess && deviceCount > 0);
}

__host__ void cpu_to_cuda(Tensor* tensor) {
    if (tensor->device && strcmp(tensor->device.get(), "cuda") == 0) {
        return;
    }

    float* device_data;
    hipError_t error = hipMalloc((void**)&device_data, tensor->size * sizeof(float));
    
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::runtime_error("Failed to allocate GPU memory");
    }

    error = hipMemcpy(device_data, tensor->data.get(), tensor->size * sizeof(float), hipMemcpyHostToDevice);
    
    if (error != hipSuccess) {
        hipFree(device_data);
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::runtime_error("Failed to copy data to GPU");
    }

    auto cuda_deleter = [](float* ptr) { hipFree(ptr); };

    tensor->data = std::shared_ptr<float[]>(device_data, cuda_deleter);

    const char* device_str = "cuda";
    size_t str_len = strlen(device_str) + 1;
    tensor->device = std::shared_ptr<char[]>(strdup(device_str), [](char* p) { free(p); });
}

__host__ void cuda_to_cpu(Tensor* tensor) {
    if (!tensor->device || strcmp(tensor->device.get(), "cpu") == 0) {
        return;
    }

    float* host_data = new float[tensor->size];

    hipError_t error = hipMemcpy(host_data, tensor->data.get(), tensor->size * sizeof(float), hipMemcpyDeviceToHost);
    
    if (error != hipSuccess) {
        delete[] host_data;
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::runtime_error("Failed to copy data from GPU");
    }

    tensor->data = std::shared_ptr<float[]>(host_data);

    const char* device_str = "cpu";
    size_t str_len = strlen(device_str) + 1;
    tensor->device = std::shared_ptr<char[]>(strdup(device_str), [](char* p) { free(p); });
}

void to_device(Tensor* tensor, const char* target_device) {
    const char* current_device = tensor->device ? tensor->device.get() : "cpu";

    if (strcmp(current_device, target_device) == 0) {
        return;
    }
    
    if (strcmp(target_device, "cuda") == 0) {
        cpu_to_cuda(tensor);
    } 
    else if (strcmp(target_device, "cpu") == 0) {
        cuda_to_cpu(tensor);
    } 
    else {
        fprintf(stderr, "Unsupported device: %s\n", target_device);
        throw std::runtime_error("Unsupported device");
    }
}

__global__ void add_kernel(const float* a, const float* b, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] + b[idx];
    }
}

Tensor add_tensor_cuda(const Tensor& a, const Tensor& b) {
    if (a.size != b.size) {
        throw std::runtime_error("Tensors must have same size for CUDA addition");
    }

    float* result_data;
    hipMalloc(&result_data, a.size * sizeof(float));

    int block_size = 256;
    int num_blocks = (a.size + block_size - 1) / block_size;
    add_kernel<<<num_blocks, block_size>>>(a.data.get(), b.data.get(), result_data, a.size);
    hipDeviceSynchronize();

    int* shape_copy = new int[a.ndim];
    memcpy(shape_copy, a.shape.get(), a.ndim * sizeof(int));

    auto cuda_deleter = [](float* ptr) { hipFree(ptr); };
    std::shared_ptr<float[]> shared_result(result_data, cuda_deleter);
    
    Tensor result(shared_result, shape_copy, a.ndim);
    
    const char* device_str = "cuda";
    size_t str_len = strlen(device_str) + 1;
    result.device = std::shared_ptr<char[]>(strdup("cuda"), [](char* p) { delete[] p; });
    
    return result;
}

__global__ void subtract_kernel(const float* a, const float* b, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] - b[idx];
    }
}

Tensor sub_tensor_cuda(const Tensor& a, const Tensor& b) {
    if (a.size != b.size) {
        throw std::runtime_error("Tensors must have same size for CUDA subtraction");
    }

    float* result_data;
    hipMalloc(&result_data, a.size * sizeof(float));

    int block_size = 256;
    int num_blocks = (a.size + block_size - 1) / block_size;
    subtract_kernel<<<num_blocks, block_size>>>(a.data.get(), b.data.get(), result_data, a.size);
    hipDeviceSynchronize();

    int* shape_copy = new int[a.ndim];
    memcpy(shape_copy, a.shape.get(), a.ndim * sizeof(int));

    auto cuda_deleter = [](float* ptr) { hipFree(ptr); };
    std::shared_ptr<float[]> shared_result(result_data, cuda_deleter);
    
    Tensor result(shared_result, shape_copy, a.ndim);
    result.device = std::shared_ptr<char[]>(strdup("cuda"), [](char* p) { delete[] p; });
    
    return result;
}

__global__ void multiply_kernel(const float* a, const float* b, float* result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] * b[idx];
    }
}

Tensor mul_tensor_cuda(const Tensor& a, const Tensor& b) {
    if (a.size != b.size) throw std::runtime_error("Tensors must have same size for CUDA multiplication");
    
    float* result_data;
    hipMalloc(&result_data, a.size * sizeof(float));
    
    int block_size = 256;
    int num_blocks = (a.size + block_size - 1) / block_size;
    multiply_kernel<<<num_blocks, block_size>>>(a.data.get(), b.data.get(), result_data, a.size);
    hipDeviceSynchronize();
    
    int* shape_copy = new int[a.ndim];
    memcpy(shape_copy, a.shape.get(), a.ndim * sizeof(int));
    
    auto deleter = [](float* p) { hipFree(p); };
    Tensor result(std::shared_ptr<float[]>(result_data, deleter), shape_copy, a.ndim);
    result.device = std::shared_ptr<char[]>(strdup("cuda"), [](char* p) { free(p); });
    
    return result;
}

__global__ void transpose_2d_kernel(const float* input, float* output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < cols && y < rows) {
        output[x * rows + y] = input[y * cols + x];
    }
}

__global__ void transpose_3d_kernel(const float* input, float* output, int batch, int rows, int cols) {
    int i = blockIdx.z;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < cols && j < rows && i < batch) {
        int input_index = i * rows * cols + j * cols + k;
        int output_index = k * rows * batch + j * batch + i;
        
        output[output_index] = input[input_index];
    }
}

std::shared_ptr<Tensor> transpose_tensor_cuda(const Tensor& tensor) {
    std::vector<int> new_shape(tensor.ndim);
    for (int i = 0; i < tensor.ndim; i++) {
        new_shape[i] = tensor.shape.get()[tensor.ndim - 1 - i];
    }

    float* d_result;
    hipMalloc(&d_result, tensor.size * sizeof(float));

    dim3 block(16, 16);
    if (tensor.ndim == 1) {
        // 1D: Direct copy
        hipMemcpy(d_result, tensor.data.get(), tensor.size * sizeof(float), hipMemcpyDeviceToDevice);
    } else if (tensor.ndim == 2) {
        int rows = tensor.shape[0], cols = tensor.shape[1];
        dim3 grid((cols + block.x - 1)/block.x, (rows + block.y - 1)/block.y);
        transpose_2d_kernel<<<grid, block>>>(tensor.data.get(), d_result, rows, cols);
    } else if (tensor.ndim == 3) {
        int batch = tensor.shape[0], rows = tensor.shape[1], cols = tensor.shape[2];
        dim3 grid(
            (cols + block.x - 1) / block.x,   // Columns become first dimension
            (rows + block.y - 1) / block.y,   // Rows remain second
            batch                             // Batch becomes third
        );
        transpose_3d_kernel<<<grid, block>>>(tensor.data.get(), d_result, batch, rows, cols);
    } else {
        hipFree(d_result);
        throw std::runtime_error("Unsupported dimension for CUDA transpose");
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_result);
        throw std::runtime_error("CUDA error: " + std::string(hipGetErrorString(err)));
    }

    hipDeviceSynchronize();

    int* shape_copy = new int[tensor.ndim];
    memcpy(shape_copy, new_shape.data(), tensor.ndim * sizeof(int));
    auto deleter = [](float* p) { hipFree(p); };
    auto result = std::make_shared<Tensor>(std::shared_ptr<float[]>(d_result, deleter), shape_copy, tensor.ndim);
    result->device = std::shared_ptr<char[]>(strdup("cuda"), [](char* p) { free(p); });
    return result;
}

#else

bool is_cuda_available() {
    return false;
}

void cpu_to_cuda(Tensor* tensor) {
    fprintf(stderr, "CUDA not available in this build\n");
    throw std::runtime_error("CUDA not available");
}

void cuda_to_cpu(Tensor* tensor) {
    fprintf(stderr, "CUDA not available in this build\n");
    throw std::runtime_error("CUDA not available");
}

void to_device(Tensor* tensor, const char* target_device) {
    if (strcmp(target_device, "cuda") == 0) {
        fprintf(stderr, "CUDA not available in this build\n");
        throw std::runtime_error("CUDA not available");
    }
}

Tensor add_tensor_cuda(const Tensor& a, const Tensor& b) {
    fprintf(stderr, "CUDA not available in this build\n");
    throw std::runtime_error("CUDA not available");
}

Tensor sub_tensor_cuda(const Tensor& a, const Tensor& b) {
    fprintf(stderr, "CUDA not available in this build\n");
    throw std::runtime_error("CUDA not available");
}

Tensor mul_tensor_cuda(const Tensor& a, const Tensor& b) {
    fprintf(stderr, "CUDA not available in this build\n");
    throw std::runtime_error("CUDA not available");
}

#endif // __HIPCC__